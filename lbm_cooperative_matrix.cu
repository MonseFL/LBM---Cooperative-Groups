#include "hip/hip_runtime.h"
﻿
#include <string>
#include <math.h>
#include <sstream>
#include <iomanip>
#include <vector>
#include <stdio.h>
#include <stdlib.h>
#include <iostream>
#include <fstream>
#include <hip/hip_runtime.h>
#include <time.h>
#include <sys/types.h> 
#include <sys/stat.h>
#include <hip/hip_cooperative_groups.h>
using namespace cooperative_groups;

#if defined(_WIN32)  
#include <direct.h> 
#endif

typedef double prec;

#define BLOCK_SIZE 25;
#define X_SIZE 5;
#define Y_SIZE 5;
#define FILE_BATCH_SIZE 1000000
#define EARTH_VEL 0.000072921

typedef struct mainHStruct {
	int* node_types;
	prec* b;
	prec* w;
	int* node_values;
	int* TSind;
	prec* TSdata;
} mainHStruct;

typedef struct mainDStruct {
	int Lx;
	int Ly;
	int NTS;
	int TTS;
	int Nblocks;
	int Ngrid;
	int Nblocks_real;
	int Ngrid_real;
	int* node_types;
	prec* b;
	prec* w;
	int* node_values;
	int* TSind;
	prec* TSdata;
} mainDStruct;

typedef struct cudaStruct {
	prec tau;
	prec g;
	prec e;
	int* ex;
	int* ey;
	unsigned char* SC_bin;
	unsigned char* BB_bin;

	prec* h;
	prec* ux;
	prec* uy;
	prec* f1;
	prec* f2;
	prec* force;
} cudaStruct;


prec stod(char* word, int len) {
	prec val = 0, ord = 1E-16;
	int dig;
	for (int i = len - 1; i >= 0; i--) {
		if (word[i] != '.') {
			dig = word[i] - '0';
			val += dig * ord;
			ord *= 10;
		}
	}
	return val;
}

void readInput(prec** b, prec** w,
	int** node_types, int** node_values, std::string inputdir,
	int* Lx, int* Ly, prec* Dx, prec* x0, prec* y0, int x_size, int y_size) {
	FILE* fp;
	std::string fullfile = inputdir + ".txt";
	if ((fp = fopen(fullfile.c_str(), "r")) == NULL) {
		std::cout << "Input file doesn't exist." << std::endl;
		exit(EXIT_FAILURE);
	}
	std::cout << "Reading input from " << fullfile << std::endl;

	fscanf(fp, "%d %d %lf %lf %lf\n", Lx, Ly, Dx, x0, y0);

	int Lx_extras = 0;
	int Ly_extras = 0;

	int H_blocks = (int)ceil(((prec)(*Lx) - (prec)x_size) / (x_size - 1)) + 1;
	int V_blocks = (int)ceil(((prec)(*Lx) - (prec)y_size) / (y_size - 1)) + 1;


	if (((*Lx) -x_size) % (x_size -1 ) != 0) {
		int bloquesX = (*Lx) / x_size;
		Lx_extras =(x_size-1)- ((*Lx) - x_size) % (x_size - 1);

	}

	if (((*Ly) - y_size) % (y_size - 1) != 0) {
		int bloquesY = (*Ly) / y_size;
		Ly_extras = (y_size - 1) - ((*Ly) - y_size) % (y_size - 1);

	}


	prec* bl = new prec[((*Lx) + Lx_extras) * ((*Ly)+ Ly_extras)];
	prec* wl = new prec[((*Lx) + Lx_extras) * ((*Ly) + Ly_extras)];
	int* node_typesl = new int[((*Lx) + Lx_extras) * ((*Ly) + Ly_extras)];
	int* node_valuesl = new int[((*Lx) + Lx_extras) * ((*Ly) + Ly_extras)];
	int wc = 0, bc = 0, len = 0, buflen;
	prec val;
	char buffer[FILE_BATCH_SIZE], word[50];
	buffer[FILE_BATCH_SIZE - 1] = '\0';

	
	while (wc < ((*Lx) + Lx_extras) * ((*Ly) + Ly_extras)) {
		buflen = fread(buffer, 1, FILE_BATCH_SIZE - 1, fp);
		for (int i = 0; i <= buflen; i++) {
			if (wc - ((wc / ((*Lx) + Lx_extras)) + 1) * (*Lx) >= 0) {
				wl[wc] = 0;
				node_typesl[wc] = 0;
				wc++;
				bl[bc] = 0;
				bc++;

			}
			else if ((wc / ((*Ly) + Ly_extras)) - (*Ly) >= 0) {
				wl[wc] = 0;
					node_typesl[wc] = 0;
					wc++;
					bl[bc] = 0;
					bc++;
			}
			else {
				if (buffer[i] == ' ' || buffer[i] == '\n' || buffer[i] == '\r') {
					word[len] = '\0';
						if (len == 1) {

							node_typesl[wc - 1] = word[0] - '0';
							//std::cout << node_typesl[wc - 1];
						}
						else if (len > 1) {
							val = stod(word, len);
							if (wc == bc) {
								bl[bc] = val;
								bc++;
							}
							else {
								wl[wc] = val;
								wc++;
							}
						}
					len = 0;
					}
				else {
					word[len] = buffer[i];
					len++;
				}

			}
		}
		if (buflen != FILE_BATCH_SIZE - 1)
			break;
		if (buffer[buflen - 1] != ' ' && buffer[buflen - 1] != '\n' && buffer[buflen - 1] != '\r')
			fseek(fp, 1 - len, SEEK_CUR);
		len = 0;
	}
	fclose(fp);
	*w = wl;
	*b = bl;
	*node_types = node_typesl;

	int contador_x = 1;
	int contador_y = 1;
	int borde_x, borde_y;
	int y, x;
	for (int i = 0; i < ((*Lx) + Lx_extras) * ((*Ly) + Ly_extras); i++) {
		y = i / ((*Lx) + Lx_extras);
		x = i % ((*Ly) + Ly_extras);
		if (x == ((*Lx) + Lx_extras) - 1) {
			contador_x = 1;

		}


		borde_x = contador_x * x_size - contador_x;
		borde_y = contador_y * y_size - contador_y;
		if (x == borde_x && y == borde_y) {
			node_valuesl[i] = 4;

			contador_x++;
		}
		else if (x == borde_x || y == borde_y) {
			node_valuesl[i] = 2;
			contador_x = (x == borde_x) ? contador_x + 1 : contador_x;
		}
		else {
			node_valuesl[i] = 1;
		}
		if (x == ((*Lx) + Lx_extras) - 1 && y == borde_y && y != ((*Ly) + Ly_extras) - y_size) {
			contador_y++;
		}

	}


	*node_values = node_valuesl;
}

//void readInput(prec** b, prec** w,
//	int** node_types,int** node_values,
//	int* Lx, int* Ly, prec* Dx, prec* x0, prec* y0, int x_size, int y_size) {
//	FILE* fp;
//	std::string fullfile = "test_63x64.txt";
//	if ((fp = fopen(fullfile.c_str(), "r")) == NULL) {
//		std::cout << "Input file doesn't exist." << std::endl;
//		exit(EXIT_FAILURE);
//	}
//	std::cout << "Reading input from " << fullfile << std::endl;
//
//	fscanf(fp, "%d %d %lf %lf %lf\n", Lx, Ly, Dx, x0, y0);
//
//	std::cout << "datos leidos\n";
//
//	prec* bl = new prec[(*Lx) * (*Ly)];
//	prec* wl = new prec[(*Lx) * (*Ly)];
//	int* node_typesl = new int[(*Lx) * (*Ly)];
//	int* node_valuesl = new int[(*Lx) * (*Ly)];
//	//int wc = 0, bc = 0, len = 0, buflen;
//	prec val;
//	int y, x;
//	int contador_x = 1;
//	int contador_y = 1;
//	int borde_x, borde_y;
//	for (int i = 0; i < (*Lx) * (*Ly); i++) {
//
//		fscanf(fp, "%lf %lf", &bl[i], &wl[i]);
//
//		y = i / (*Lx);
//		x = i % (*Lx);
//		if (x == (*Lx) - 1) {
//			node_typesl[i] = 2;
//		}
//		else if (x == 0 || y == 0 || y == (*Ly) - 1) {
//			node_typesl[i] = 2;
//		}
//		else {
//			node_typesl[i] = 0;
//		}
//
//		if (x == (*Lx) - 1) {
//			contador_x = 1;
//
//		}
//
//		
//		borde_x = contador_x * x_size - contador_x;
//		borde_y = contador_y * y_size - contador_y;
//		if (x ==borde_x && y == borde_y ) {
//			node_valuesl[i] = 4;
//
//			contador_x++;
//		}
//		else if (x == borde_x || y == borde_y) {
//			node_valuesl[i] = 2;
//			contador_x = (x == borde_x) ? contador_x + 1 : contador_x;
//		}
//		else {
//			node_valuesl[i] = 1;
//		}
//		if (x == (*Lx) - 1 && y == borde_y && y != (*Ly) - y_size) {
//			contador_y++;
//		}
//		
//	}
//	std::cout << "arreglo iniciado";
//	fclose(fp);
//	*w = wl;
//	*b = bl;
//	*node_types = node_typesl;
//	*node_values = node_valuesl;
//}







void freemem(mainHStruct host, mainDStruct devi, cudaStruct devEx) {
	/*delete[] host.b;
	delete[] host.w;
	delete[] host.ux;
	delete[] host.uy;
	delete[] host.node_types;
	*/

	hipFree(devi.b);
	hipFree(devi.w);
	hipFree(devi.node_types);
	hipFree(devi.TSind);
	hipFree(devi.TSdata);

	hipFree(devEx.ex);
	hipFree(devEx.ey);
	hipFree(devEx.h);
	hipFree(devEx.f1);
	hipFree(devEx.f2);
#if IN == 3
	hipFree(devEx.Arr_tri);
#elif IN == 4
	hipFree(devEx.SC_bin);
	hipFree(devEx.BB_bin);
#endif
}






__global__ void LBMpush(int Lx, int Ly, prec g, prec e, prec tau, const unsigned char* __restrict__ SC_bin,
	const unsigned char* __restrict__ BB_bin, int* node_value, const prec* __restrict__ f1,
	prec* f2, prec* h, prec* b, prec* ux, prec* uy, int x_size, int y_size, int block_size, int H_blocks, int t) {

	int i = threadIdx.x + blockIdx.x * blockDim.x;
	int size = Lx * Ly;
	thread_block block_group = this_thread_block();


	int neigh[8];
	prec ftemp[9], ftemp2[9];
	prec source[8];
	prec feq[9];

	prec factor = 1 / (6 * e * e);
	prec factor21 = -(1 * g / 3.0) / (e * e);
	prec factor22 = -(1 * g / 12.0) / (e * e);
	prec tauVal = tau - 1;


	prec ux3, uy3, uxuy5, uxuy6;
	prec fact1 = 1 / (9 * e * e);
	prec fact2 = fact1 * 0.25;

	prec gh, usq;
	unsigned char SC, BB;



	// calculo de indices
	int x_local = i % x_size;
	int y_local = i % block_size / x_size;

	int y = y_local + (blockIdx.x / H_blocks) * y_size - (blockIdx.x / H_blocks);
	int x = x_local + (blockIdx.x % H_blocks) * x_size - (blockIdx.x % H_blocks);

	int i_read = x + y * Lx;

	SC = SC_bin[i_read];
	BB = BB_bin[i_read];

	if (SC + BB != 0) {

		//calculo vecinos
		neigh[0] = (x != Lx - 1) ? i_read + 1 : i_read;
		neigh[1] = (y != 0) ? i_read - Lx : i_read;
		neigh[2] = (x != 0) ? i_read - 1 : i_read;
		neigh[3] = (y != Ly - 1) ? i_read + Lx : i_read;
		neigh[4] = (y != 0 && x != Lx - 1) ? i_read - Lx + 1 : i_read;
		neigh[5] = (y != 0 && x != 0) ? i_read - Lx - 1 : i_read;
		neigh[6] = (y != Ly - 1 && x != 0) ? i_read + Lx - 1 : i_read;
		neigh[7] = (y != Ly - 1 && x != Lx - 1) ? i_read + Lx + 1 : i_read;

		//Variables locales

		prec localh = h[i_read];
		prec localux = ux[i_read];
		prec localuy = uy[i_read];




		//Source term
		source[0] = factor21 * 0.5 * (h[neigh[0]] + localh) * (b[neigh[0]] - b[i_read]);
		source[1] = factor21 * 0.5 * (h[neigh[1]] + localh) * (b[neigh[1]] - b[i_read]);
		source[2] = factor21 * 0.5 * (h[neigh[2]] + localh) * (b[neigh[2]] - b[i_read]);
		source[3] = factor21 * 0.5 * (h[neigh[3]] + localh) * (b[neigh[3]] - b[i_read]);
		source[4] = factor22 * 0.5 * (h[neigh[4]] + localh) * (b[neigh[4]] - b[i_read]);
		source[5] = factor22 * 0.5 * (h[neigh[5]] + localh) * (b[neigh[5]] - b[i_read]);
		source[6] = factor22 * 0.5 * (h[neigh[6]] + localh) * (b[neigh[6]] - b[i_read]);
		source[7] = factor22 * 0.5 * (h[neigh[7]] + localh) * (b[neigh[7]] - b[i_read]);





		gh = 1.5 * g * localh;
		usq = 1.5 * (localux * localux + localuy * localuy);
		ux3 = 3.0 * e * localux;
		uy3 = 3.0 * e * localuy;
		uxuy5 = ux3 + uy3;
		uxuy6 = uy3 - ux3;

		feq[0] = localh - fact1 * localh * (5.0 * gh + 4.0 * usq);
		feq[1] = fact1 * localh * (gh + ux3 + 0.5 * ux3 * ux3 * 9 * fact1 - usq);
		feq[2] = fact1 * localh * (gh + uy3 + 0.5 * uy3 * uy3 * 9 * fact1 - usq);
		feq[3] = fact1 * localh * (gh - ux3 + 0.5 * ux3 * ux3 * 9 * fact1 - usq);
		feq[4] = fact1 * localh * (gh - uy3 + 0.5 * uy3 * uy3 * 9 * fact1 - usq);
		feq[5] = fact2 * localh * (gh + uxuy5 + 0.5 * uxuy5 * uxuy5 * 9 * fact1 - usq);
		feq[6] = fact2 * localh * (gh + uxuy6 + 0.5 * uxuy6 * uxuy6 * 9 * fact1 - usq);
		feq[7] = fact2 * localh * (gh - uxuy5 + 0.5 * uxuy5 * uxuy5 * 9 * fact1 - usq);
		feq[8] = fact2 * localh * (gh - uxuy6 + 0.5 * uxuy6 * uxuy6 * 9 * fact1 - usq);


		//Streaming
		f2[i_read] = (f1[i_read] * tauVal + feq[0]) / tau;
		if (neigh[0] != i_read) f2[neigh[0] + size] = (f1[i_read + size] * tauVal + feq[1]) / tau + source[0];
		if (neigh[1] != i_read) f2[neigh[1] + 2 * size] = (f1[i_read + 2 * size] * tauVal + feq[2]) / tau + source[1];
		if (neigh[2] != i_read) f2[neigh[2] + 3 * size] = (f1[i_read + 3 * size] * tauVal + feq[3]) / tau + source[2];
		if (neigh[3] != i_read) f2[neigh[3] + 4 * size] = (f1[i_read + 4 * size] * tauVal + feq[4]) / tau + source[3];
		if (neigh[4] != i_read) f2[neigh[4] + 5 * size] = (f1[i_read + 5 * size] * tauVal + feq[5]) / tau + source[4];
		if (neigh[5] != i_read) f2[neigh[5] + 6 * size] = (f1[i_read + 6 * size] * tauVal + feq[6]) / tau + source[5];
		if (neigh[6] != i_read) f2[neigh[6] + 7 * size] = (f1[i_read + 7 * size] * tauVal + feq[7]) / tau + source[6];
		if (neigh[7] != i_read) f2[neigh[7] + 8 * size] = (f1[i_read + 8 * size] * tauVal + feq[8]) / tau + source[7];



		block_group.sync();


		int old = atomicAdd(&node_value[i_read], 1);
		int type = old / t;
		old = (type < 4) ? old % 2 : 4 - old % 4;
		if (old == 1 || type == 1) {



			ftemp2[0] = f2[i_read];
			ftemp2[1] = f2[i_read + size];
			ftemp2[2] = f2[i_read + 2 * size];
			ftemp2[3] = f2[i_read + 3 * size];
			ftemp2[4] = f2[i_read + 4 * size];
			ftemp2[5] = f2[i_read + 5 * size];
			ftemp2[6] = f2[i_read + 6 * size];
			ftemp2[7] = f2[i_read + 7 * size];
			ftemp2[8] = f2[i_read + 8 * size];




			if ((SC >> 0) & 1) ftemp2[1] = (f1[i_read + size] * tauVal + feq[1]) / tau;
			if ((SC >> 1) & 1) ftemp2[2] = (f1[i_read + 2 * size] * tauVal + feq[2]) / tau;
			if ((SC >> 2) & 1) ftemp2[3] = (f1[i_read + 3 * size] * tauVal + feq[3]) / tau;
			if ((SC >> 3) & 1) ftemp2[4] = (f1[i_read + 4 * size] * tauVal + feq[4]) / tau;
			if ((SC >> 4) & 1) ftemp2[5] = (f1[i_read + 5 * size] * tauVal + feq[5]) / tau;
			if ((SC >> 5) & 1) ftemp2[6] = (f1[i_read + 6 * size] * tauVal + feq[6]) / tau;
			if ((SC >> 6) & 1) ftemp2[7] = (f1[i_read + 7 * size] * tauVal + feq[7]) / tau;
			if ((SC >> 7) & 1) ftemp2[8] = (f1[i_read + 8 * size] * tauVal + feq[8]) / tau;

			ftemp2[1] = ((BB >> (0)) & 1) ? f1[i_read + 3 * size] : ftemp2[1];
			ftemp2[2] = ((BB >> (1)) & 1) ? f1[i_read + 4 * size] : ftemp2[2];
			ftemp2[3] = ((BB >> (2)) & 1) ? f1[i_read + 1 * size] : ftemp2[3];
			ftemp2[4] = ((BB >> (3)) & 1) ? f1[i_read + 2 * size] : ftemp2[4];
			ftemp2[5] = ((BB >> (4)) & 1) ? f1[i_read + 7 * size] : ftemp2[5];
			ftemp2[6] = ((BB >> (5)) & 1) ? f1[i_read + 8 * size] : ftemp2[6];
			ftemp2[7] = ((BB >> (6)) & 1) ? f1[i_read + 5 * size] : ftemp2[7];
			ftemp2[8] = ((BB >> (7)) & 1) ? f1[i_read + 6 * size] : ftemp2[8];



			f2[i_read + size] = ftemp2[1];
			f2[i_read + 2 * size] = ftemp2[2];
			f2[i_read + 3 * size] = ftemp2[3];
			f2[i_read + 4 * size] = ftemp2[4];
			f2[i_read + 5 * size] = ftemp2[5];
			f2[i_read + 6 * size] = ftemp2[6];
			f2[i_read + 7 * size] = ftemp2[7];
			f2[i_read + 8 * size] = ftemp2[8];

			h[i_read] = ftemp2[0] + (ftemp2[1] + ftemp2[2] + ftemp2[3] + ftemp2[4]) + (ftemp2[5] + ftemp2[6] + ftemp2[7] + ftemp2[8]);
			ux[i_read] = e * ((ftemp2[1] - ftemp2[3]) + (ftemp2[5] - ftemp2[6] - ftemp2[7] + ftemp2[8])) / h[i_read];
			uy[i_read] = e * ((ftemp2[2] - ftemp2[4]) + (ftemp2[5] + ftemp2[6] - ftemp2[7] - ftemp2[8])) / h[i_read];

			//calculo variables macroscopicas




		}

	}
}




//__global__ void auxArraysKernel(int Lx, int Ly,
//	const int* __restrict__ ex, const int* __restrict__ ey,
//	const int* __restrict__ node_types,
//	unsigned char* SC_bin, unsigned char* BB_bin) {
//
//
//
//	int i = threadIdx.x + blockIdx.x * blockDim.x;
//
//
//	int size = Lx * Ly;
//	if (i < size) {
//		int y = (int)i / Lx;
//		int x = i - y * Lx;
//		//int xi, yi, ind, indj, indk, a;
//		int valueSC = 0, valueBB = 0;
//		if (node_types[i] == 1) {
//			valueBB = 64 + 32 + 8;
//			if (y == 0) {
//				valueSC = 8 + 128;
//			}
//			if (y == Ly - 1) {
//				valueSC = 2 + 16;
//			}
//		}
//		else if (node_types[i] == 2) {
//			if (y == 0) {
//				valueSC = 8 + 64 + 128;
//				if (x == 0) {
//					valueSC = 1 + 8 + 16 + 64 + 128;
//				}
//				if (x == Lx - 1) {
//					valueSC = 4 + 8 + 32 + 64 + 128;
//				}
//			}
//			else if (y == Ly - 1) {
//				valueSC = 2 + 16 + 32;
//				if (x == 0) {
//					valueSC = 1 + 2 + 16 + 32 + 128;
//				}
//				if (x == Lx - 1) {
//					valueSC = 2 + 4 + 16 + 32 + 64;
//				}
//
//			}
//			else if (x == 0) {
//				valueSC = 128 + 16 + 1;
//			}
//			else if (x == Lx - 1) {
//				valueSC = 4 + 32 + 64;
//			}
//
//		}
//
//		SC_bin[i] = (unsigned char)valueSC;
//		BB_bin[i] = (unsigned char)valueBB;
//	}
//}

__global__ void auxArraysKernel(int Lx, int Ly, int Lxo, int Lyo,
	const int* __restrict__ ex, const int* __restrict__ ey,
	const int* __restrict__ node_types,
	unsigned char* SC_bin, unsigned char* BB_bin) {

	int i = threadIdx.x + blockIdx.x * blockDim.x;
	int size = Lx * Ly;
	if (i < size) {
		int y = (int)i / Lx;
		int x = i - y * Lx;
		int xi, yi, ind, indj, indk, a;
		int valueSC = 0, valueBB = 0;
		if (node_types[i] == 2) {
			if (y == 0) {
				if (x == 0)
					valueSC += 1 + 8 + 16 + 64 + 128;
				else if (x == Lxo - 1)
					valueSC += 4 + 8 + 32 + 64 + 128;
				else
					valueSC += 8 + 64 + 128;
			}
			else if (y == Lyo - 1) {
				if (x == 0)
					valueSC += 1 + 2 + 16 + 32 + 128;
				else if (x == Lxo - 1)
					valueSC += 2 + 4 + 16 + 32 + 64;
				else
					valueSC += 2 + 16 + 32;
			}
			else {
				if (x == 0)
					valueSC += 1 + 16 + 128;
				else if (x == Lxo - 1)
					valueSC += 4 + 32 + 64;
				else
					valueSC = 0;
			}
		}
		else if (node_types[i] == 1) {
			if (y == 0) {
				if (x == 0) {
					valueBB += 1 + 8 + 128;
					valueSC += 16 + 64;
				}
				else if (x == Lxo - 1) {
					valueBB += 4 + 8 + 64;
					valueSC += 32 + 128;
				}
				else {
					valueBB += 8 + 64 + 128;
				}
			}
			else if (y == Lyo - 1) {
				if (x == 0) {
					valueBB += 1 + 2 + 16;
					valueSC += 32 + 128;
				}
				else if (x == Lxo - 1) {
					valueBB += 2 + 4 + 32;
					valueSC += 16 + 64;
				}
				else {
					valueBB += 2 + 16 + 32;
				}

			}
			else {
				if (x == 0)
					valueBB += 1 + 16 + 128;
				else if (x == Lx - 1)
					valueBB += 4 + 32 + 64;
				else
					valueBB = 0;
			}

		}
		else {
			valueBB += 0;
			valueSC += 0;
		}
		SC_bin[i] = (unsigned char)valueSC;
		BB_bin[i] = (unsigned char)valueBB;
	}
}

__global__ void hKernel(int Lx, int Ly, const prec* __restrict__ w,
	const prec* __restrict__ b, prec* h) {

	int i = threadIdx.x + blockIdx.x * blockDim.x;
	if (i < Lx * Ly) {
		h[i] = w[i] - b[i];
	}
}

__global__ void feqKernel(int Lx, int Ly, prec g, prec e,
	const prec* __restrict__ h, prec* f) {

	int i = threadIdx.x + blockIdx.x * blockDim.x;
	if (i < Lx * Ly) {
		prec hi = h[i];
		prec gh1 = g * hi * hi / (6.0 * e * e);
		prec gh2 = gh1 / 4;
		//printf("%d %f %f %f %f\n", i, gh1, hi, g, e);
		f[i] = hi - 5.0 * gh1;
		//printf("%f\n", f[i]);
		f[i + (Lx * Ly)] = gh1;
		f[i + (2 * Lx * Ly)] = gh1;
		f[i + (3 * Lx * Ly)] = gh1;
		f[i + (4 * Lx * Ly)] = gh1;
		f[i + (5 * Lx * Ly)] = gh2;
		f[i + (6 * Lx * Ly)] = gh2;
		f[i + (7 * Lx * Ly)] = gh2;
		f[i + (8 * Lx * Ly)] = gh2;
	}
}

void setup(mainDStruct devi, cudaStruct devEx, int x_size, int y_size, int H_blocks, int Lxo, int Lyo) {

	hipError_t err = hipGetLastError();
	printf("\n aux init\n");
	auxArraysKernel << <devi.Ngrid_real, devi.Nblocks >> > (devi.Lx, devi.Ly,Lxo,Lyo, devEx.ex, devEx.ey, devi.node_types,
		devEx.SC_bin, devEx.BB_bin);
	if (err != hipSuccess)
	{
		printf("CUDA Error: %s\n", hipGetErrorString(err));
		exit(-1);
	}
	printf("\nauxArraysKernel finish\n");

	hKernel << <devi.Ngrid_real, devi.Nblocks >> > (devi.Lx, devi.Ly, devi.w, devi.b, devEx.h);
	if (err != hipSuccess)
	{
		printf("CUDA Error: %s\n", hipGetErrorString(err));
		exit(-1);
	}
	printf("\n hKernel finish\n");
	printf("\n feqKernel init\n");

	feqKernel << <devi.Ngrid_real, devi.Nblocks >> > (devi.Lx, devi.Ly, devEx.g, devEx.e, devEx.h, devEx.f1);

	if (err != hipSuccess)
	{
		printf("CUDA Error: %s\n", hipGetErrorString(err));
		exit(-1);
	}

	printf("\n feqKernel finish\n");

	//TSkernel << <devi.NTS, 1 >> > (devi.TSdata, devi.w, devi.TSind, 0, deltaTS, devi.NTS, devi.TTS);
}


__global__ void wKernel(int Lx, int Ly, const prec* __restrict__ h,
	const prec* __restrict__ b, prec* w) {

	int i = threadIdx.x + blockIdx.x * blockDim.x;
	if (i < Lx * Ly) {
		w[i] = h[i] + b[i];
	}
}

__global__ void orderKernel(int Lx, int Ly, int H_blocks, int y_size, int x_size, const prec* __restrict__ h,
	const prec* __restrict__ b, prec* w) {

	int i = threadIdx.x + blockIdx.x * blockDim.x;

	int x_local = i % x_size;
	int y_local = i / x_size - blockIdx.x * y_size;


	int y = y_local + (blockIdx.x / H_blocks) * y_size;
	int x = x_local + (blockIdx.x % H_blocks) * x_size;

	int ind = x + y * x_size;

	if (i < Lx * Ly) {
		w[i] = h[ind] + b[ind];
	}


}

void writeOutput(int L, int t, prec* w, std::string outputdir) {
	FILE* fp;
	std::ostringstream numero;
	numero << std::setw(5) << std::setfill('0') << std::right << (t);
	std::string fullfile = outputdir + ".dat";
	if ((fp = fopen(fullfile.c_str(), "wb")) == NULL) {
		std::cout << "Can't create output file." << std::endl;
		exit(EXIT_FAILURE);
	}
	fwrite(&w[0], sizeof(prec), L, fp);
	fclose(fp);
}
void writeOutput2(int L, int t, int* w, std::string outputdir) {
	FILE* fp;
	std::ostringstream numero;
	numero << std::setw(5) << std::setfill('0') << std::right << (t);
	std::string fullfile = outputdir + ".dat";
	if ((fp = fopen(fullfile.c_str(), "wb")) == NULL) {
		std::cout << "Can't create output file." << std::endl;
		exit(EXIT_FAILURE);
	}
	fwrite(&w[0], sizeof(int), L, fp);
	fclose(fp);
}
void copyAndWriteResultData(mainHStruct host, mainDStruct devi, cudaStruct devEx, int t, std::string outputdir) {

	wKernel << <devi.Ngrid, devi.Nblocks >> > (devi.Lx, devi.Ly, devEx.h, devi.b, devi.w);


	hipMemcpy(host.w, devi.w, devi.Lx * devi.Ly * sizeof(prec), hipMemcpyDeviceToHost);

	writeOutput(devi.Lx * devi.Ly, t, host.w, outputdir);

}

void copyAndWriteResultData_v2(mainHStruct host, mainDStruct devi, cudaStruct devEx, int t, std::string outputdir) {

	wKernel << <devi.Ngrid, devi.Nblocks >> > (devi.Lx, devi.Ly, devEx.h, devi.b, devi.w);


	hipMemcpy(host.w, devi.w, devi.Lx * devi.Ly * sizeof(prec), hipMemcpyDeviceToHost);

	writeOutput(devi.Lx * devi.Ly, t, host.w, outputdir);

}




int main() {

	int Lx, Ly, Nblocks;
	prec Dx, x0, y0, tau, g, Dt;


	int x_size, iter;



	std::string inputdir;
	std::string outputdir = "output_SB_";

	std::cout << "Test file : ";
	std::cin >> inputdir;
	std::cout << "Block size : ";
	std::cin >> Nblocks;
	std::cout << "Group width : ";
	std::cin >> x_size;
	std::cout << "Dt : ";
	std::cin >> Dt;
	std::cout << "tau : ";
	std::cin >> tau;
	std::cout << "g : ";
	std::cin >> g;
	std::cout << "iter: ";
	std::cin >> iter;


	outputdir = outputdir + inputdir;

	mainHStruct host;
	mainDStruct devi;
	cudaStruct devEx;

	int block_size = Nblocks;
	int y_size = Nblocks / x_size;



	readInput(&host.b, &host.w, &host.node_types, &host.node_values, inputdir, &Lx, &Ly, &Dx, &x0, &y0, x_size, y_size);


	//writeOutput2(Lx * Ly, 1, host.node_values, "node_values");
	int Lxo = Lx;
	int Lyo = Ly;

	if (((Lx) - x_size) % (x_size - 1) != 0) {
		int bloquesX = (Lx) / x_size;
		int Lx_extras = (x_size - 1) - ((Lx) - x_size) % (x_size - 1);
		Lx = Lx + Lx_extras;

	}

	if (((Ly) - y_size) % (y_size - 1) != 0) {
		int bloquesY = (Ly) / y_size;
		int Ly_extras = (y_size - 1) - ((Ly) - y_size) % (y_size - 1);
		Ly = Ly + Ly_extras;

	}
	int H_blocks = (int)ceil(((prec)Lx - (prec)x_size) / (x_size - 1)) + 1;
	int V_blocks = (int)ceil(((prec)Ly - (prec)y_size) / (y_size - 1)) + 1;

	int inter_x = H_blocks - 1;
	int inter_y = V_blocks - 1;

	int s = Ly * Lx;


	std::cout << "input leido\n";

	double number;


	int num_bytes_d = Lx * Ly * sizeof(prec);

	int num_bytes_i = Lx * Ly * sizeof(int);

	int Ngrid = int(ceil((prec)(Lx + inter_x) * (prec)(Ly + inter_y) / (prec)Nblocks));
	int Ngrid_real = int(ceil((prec)Lx * (prec)Ly / (prec)Nblocks));
	int ex[9] = { 0, 1, 0,-1, 0, 1,-1,-1, 1 };
	int ey[9] = { 0, 0, 1, 0,-1, 1, 1,-1,-1 };
	prec e = Dx / Dt;

	devi.Lx = Lx;
	devi.Ly = Ly;
	devi.Nblocks = Nblocks;
	devi.Ngrid = Ngrid;

	devi.Ngrid_real = Ngrid;




	std::cout << devi.Lx << "\n";
	std::cout << devi.Ly << "\n";
	std::cout << devi.Nblocks << "\n";
	std::cout << devi.Ngrid << "\n";

	hipMalloc((void**)&devi.w, num_bytes_d);
	hipMalloc((void**)&devi.b, num_bytes_d);
	hipMalloc((void**)&devi.node_types, num_bytes_i);
	hipMalloc((void**)&devi.node_values, num_bytes_i);



	hipMemcpy(devi.b, host.b, num_bytes_d, hipMemcpyHostToDevice);
	hipMemcpy(devi.w, host.w, num_bytes_d, hipMemcpyHostToDevice);
	hipMemcpy(devi.node_types, host.node_types, num_bytes_i, hipMemcpyHostToDevice);
	hipMemcpy(devi.node_values, host.node_values, num_bytes_i, hipMemcpyHostToDevice);


	// hipMemcpy(devi.TSind, host.TSind, NTS * sizeof(int), hipMemcpyHostToDevice);

	devEx.tau = tau;
	devEx.g = g;
	devEx.e = e;
	hipMalloc((void**)&devEx.ex, 9 * sizeof(int));
	hipMalloc((void**)&devEx.ey, 9 * sizeof(int));
	hipMalloc((void**)&devEx.h, num_bytes_d);
	hipMalloc((void**)&devEx.f1, 9 * num_bytes_d);
	hipMalloc((void**)&devEx.f2, 9 * num_bytes_d);
	hipMalloc((void**)&devEx.ux, num_bytes_d);
	hipMalloc((void**)&devEx.uy, num_bytes_d);




	// #if IN == 3
	// 	hipMalloc((void**)&devEx.Arr_tri, 9 * Lx * Ly * sizeof(unsigned char));
	// #elif IN == 4
	hipMalloc((void**)&devEx.SC_bin, Lx * Ly * sizeof(unsigned char));
	hipMalloc((void**)&devEx.BB_bin, Lx * Ly * sizeof(unsigned char));


	// #endif




	int nbloquesDev;

	hipMemcpy(devEx.ex, ex, 9 * sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(devEx.ey, ey, 9 * sizeof(int), hipMemcpyHostToDevice);



	clock_t t1, t2;

	size_t a = 0;

	hipOccupancyMaxActiveBlocksPerMultiprocessor(&nbloquesDev, hKernel, 256, a);


	//std::cout << "resultado:" << nbloquesDev << std::endl;



	int t = 1; //iteraciones
	hipEvent_t ct1, ct2;
	hipEventCreate(&ct1);
	hipEventCreate(&ct2);

	float msecs = 0;
	hipDeviceSynchronize();
	setup(devi, devEx, x_size, y_size, H_blocks, Lxo, Lyo);
	std::cout << std::fixed << std::setprecision(1);
	hipEventRecord(ct1);




	std::string output_file;
	for (int i = 1; i < iter; i++) {

		if (i % 2 != 0) {
			LBMpush << < devi.Ngrid, devi.Nblocks >> > (devi.Lx, devi.Ly, devEx.g, devEx.e, devEx.tau, devEx.SC_bin, devEx.BB_bin, devi.node_values, devEx.f1, devEx.f2, devEx.h, devi.b, devEx.ux, devEx.uy, x_size, y_size, block_size, H_blocks, i);

		}
		else {
			LBMpush << < devi.Ngrid, devi.Nblocks >> > (devi.Lx, devi.Ly, devEx.g, devEx.e, devEx.tau, devEx.SC_bin, devEx.BB_bin, devi.node_values, devEx.f2, devEx.f1, devEx.h, devi.b, devEx.ux, devEx.uy, x_size, y_size, block_size, H_blocks, i);

		}
		hipError_t err = hipGetLastError();       // Get error code

		if (err != hipSuccess)
		{
			printf("CUDA Error: %s\n", hipGetErrorString(err));
			exit(-1);
		}


		/*hipMemcpy(devi.node_values, host.node_values, num_bytes_i, hipMemcpyHostToDevice);
		hipDeviceSynchronize();
		if ((i - 1) % 500 == 0) {
			hipMemcpy(host.w, devEx.h, devi.Lx * devi.Ly * sizeof(prec), hipMemcpyDeviceToHost);
			hipDeviceSynchronize();
			output_file = outputdir + "_" + std::to_string(i - 1);
			writeOutput(Lx * Ly, iter, host.w, output_file);
			 for (int i = 0; i < ((Lx)) * ((Ly)); i++) {
			 	if (i % ((Lx)) == 0) {
					std::cout << "\n";
				}
			 	printf("%f ", host.w[i]);

			 }
		}*/


	}

	std::cout << " resultado\n";




	hipEventRecord(ct2);
	hipEventSynchronize(ct2);
	hipEventElapsedTime(&msecs, ct1, ct2);
	std::cout << std::endl << "Tiempo total: " << msecs << "[ms]" << std::endl;
	std::cout << std::endl << "Tiempo promedio por iteracion: " << msecs / iter << "[ms]" << std::endl;

	int sharedBytes = 0;

	hipFree(devi.b);
	hipFree(devi.w);
	hipFree(devi.node_types);
	hipFree(devi.node_values);
	hipFree(devi.TSind);
	hipFree(devi.TSdata);

	hipFree(devEx.ex);
	hipFree(devEx.ey);
	hipFree(devEx.h);
	hipFree(devEx.f1);
	hipFree(devEx.f2);
	hipFree(devEx.SC_bin);
	hipFree(devEx.BB_bin);









	return 0;
}